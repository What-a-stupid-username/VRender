#include "hip/hip_runtime.h"
#pragma intersect intersect
#pragma bound bounds

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, plane, , );
rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, anchor, , );
rtDeclareVariable(int, lgt_instance, , ) = {0};

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx)
{
  float3 n = make_float3( plane );
  float dt = dot(ray.direction, n );
  float t = (plane.w - dot(n, ray.origin))/dt;
  if( t > ray.tmin && t < ray.tmax ) {
    float3 p = ray.origin + ray.direction * t;
    float3 vi = p - anchor;
    float a1 = dot(v1, vi);
    if(a1 >= 0 && a1 <= 1){
      float a2 = dot(v2, vi);
      if(a2 >= 0 && a2 <= 1){
        if( rtPotentialIntersection( t ) ) {
          shading_normal = geometric_normal = n;
          texcoord = make_float3(a1,a2,0);
          lgt_idx = lgt_instance;
          rtReportIntersection( 0 );
        }
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  // v1 and v2 are scaled by 1./length^2.  Rescale back to normal for the bounds computation.
  const float3 tv1  = v1 / dot( v1, v1 );
  const float3 tv2  = v2 / dot( v2, v2 );
  const float3 p00  = anchor;
  const float3 p01  = anchor + tv1;
  const float3 p10  = anchor + tv2;
  const float3 p11  = anchor + tv1 + tv2;
  const float  area = length(cross(tv1, tv2));
  
  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( p00, p01 ), fminf( p10, p11 ) );
    aabb->m_max = fmaxf( fmaxf( p00, p01 ), fmaxf( p10, p11 ) );
  } else {
    aabb->invalidate();
  }
}

