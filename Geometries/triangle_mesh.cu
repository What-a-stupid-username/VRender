#include "hip/hip_runtime.h"
#pragma bound mesh_bounds
#pragma intersect mesh_intersect_refine


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "intersection_refinement.h"

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3>   v_index_buffer;
rtBuffer<int3>   n_index_buffer;
rtBuffer<int3>   t_index_buffer;
rtBuffer<int>    material_buffer;

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


template<bool DO_REFINE>
static __device__
void meshIntersect(int primIdx)
{
	const int3 v_idx = v_index_buffer[primIdx];

	const float3 p0 = vertex_buffer[v_idx.x];
	const float3 p1 = vertex_buffer[v_idx.y];
	const float3 p2 = vertex_buffer[v_idx.z];

	// Intersect ray with triangle
	float3 n;
	float  t, beta, gamma;
	if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {

		if (rtPotentialIntersection(t)) {

			geometric_normal = normalize(n);
			if (normal_buffer.size() == 0) {
				shading_normal = geometric_normal;
			}
			else {
				const int3 n_idx = n_index_buffer[primIdx];
				float3 n0 = normal_buffer[n_idx.x];
				float3 n1 = normal_buffer[n_idx.y];
				float3 n2 = normal_buffer[n_idx.z];
				shading_normal = normalize(n1*beta + n2 * gamma + n0 * (1.0f - beta - gamma));
			}

			if (texcoord_buffer.size() == 0) {
				texcoord = make_float3(0.0f, 0.0f, 0.0f);
			}
			else {
				const int3 t_idx = t_index_buffer[primIdx];
				float2 t0 = texcoord_buffer[t_idx.x];
				float2 t1 = texcoord_buffer[t_idx.y];
				float2 t2 = texcoord_buffer[t_idx.z];
				texcoord = make_float3(t1*beta + t2 * gamma + t0 * (1.0f - beta - gamma));
			}

			if (DO_REFINE) {
				refine_and_offset_hitpoint(
					ray.origin + t * ray.direction,
					ray.direction,
					geometric_normal,
					p0,
					back_hit_point,
					front_hit_point);
			}
			int mat_index = 0;
			if (material_buffer.size() != 0)
				mat_index = material_buffer[primIdx];

			rtReportIntersection(mat_index);
		}
	}
}


RT_PROGRAM void mesh_intersect(int primIdx)
{
	meshIntersect<false>(primIdx);
}


RT_PROGRAM void mesh_intersect_refine(int primIdx)
{
	meshIntersect<true>(primIdx);
}


RT_PROGRAM void mesh_bounds(int primIdx, float result[6])
{
	const int3 v_idx = v_index_buffer[primIdx];

	const float3 v0 = vertex_buffer[v_idx.x];
	const float3 v1 = vertex_buffer[v_idx.y];
	const float3 v2 = vertex_buffer[v_idx.z];
	const float  area = length(cross(v1 - v0, v2 - v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else {
		aabb->invalidate();
	}
}

