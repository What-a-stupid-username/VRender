#include "hip/hip_runtime.h"
#pragma 0 ClosestHit
#pragma 1 AnyHit

#include "path_tracer.cuh"



//-----------------------------------------------------------------------------
//
//  default_light closest-hit
//
//-----------------------------------------------------------------------------


rtDeclareVariable(int, light_id, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
RT_PROGRAM void default_light_ClosestHit() //ray-type = 0(normal_ray)
{
	if (current_prd.depth == 0) current_prd.id = object_id;
	float3 emission_color = lights[light_id].emission;
	if (dot(ray.direction, geometric_normal) > 0)
		current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
}

RT_PROGRAM void default_light_AnyHit() //ray-type = 1(shaodw_ray)
{
	rtIgnoreIntersection();
}