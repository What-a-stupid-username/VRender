#include "hip/hip_runtime.h"
#include "path_tracer.cuh"

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, cut_off_high_variance_result, , );


RT_PROGRAM void path_tracer_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x + rnd_seed, frame_number);
    do 
    {
        //
        // Sample pixel using jittering
        //
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
		
		float3 ray_origin = eye/* + eye_jit*/;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.countEmitted = true;
        prd.seed = seed;
		prd.depth = 0;
		prd.radiance = make_float3(0);

        Ray ray = make_Ray(ray_origin, ray_direction, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_object, ray, prd);

		if (cut_off_high_variance_result) {
			float sat = 50;
			result += make_float3(min(prd.radiance.x, sat), min(prd.radiance.y, sat), min(prd.radiance.z, sat));
		}
		else {
			result += prd.radiance;
		}

    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

	//output_buffer[launch_index] = make_float4(0,0,0,1);
    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}

//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
    current_prd.radiance = bg_color;
}


