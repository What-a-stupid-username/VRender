#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"

#include "PBS.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 radiance;
    unsigned int seed;
    int depth;
    int countEmitted;
};

struct PerRayData_pathtrace_shadow
{
    float inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  rnd_seed, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  common_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;


RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x + rnd_seed, frame_number);
    do 
    {
        //
        // Sample pixel using jittering
        //
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
		
		float3 ray_origin = eye/* + eye_jit*/;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.countEmitted = true;
        prd.seed = seed;
		prd.depth = 0;
		prd.radiance = make_float3(0);

        Ray ray = make_Ray(ray_origin, ray_direction, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_object, ray, prd);

		float sat = 50;
		result += make_float3(min(prd.radiance.x,sat), min(prd.radiance.y, sat), min(prd.radiance.z, sat));


    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

	//output_buffer[launch_index] = make_float4(0,0,0,1);
    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}


//-----------------------------------------------------------------------------
//
//  default_light closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void default_light_closest_hit() //ray-type = 0(normal_ray)
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
}


//-----------------------------------------------------------------------------
//
//  default_lit_ closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,		albedo, , );
rtDeclareVariable(float,		transparent, , ) = 0.f;
rtDeclareVariable(float,		metallic, , ) = 0.f;
rtDeclareVariable(float,		smoothness, , ) = 0.f;
rtDeclareVariable(float,		refraction_index, , ) = 1.5f;


rtDeclareVariable(float3,		geometric_normal,	attribute geometric_normal, );
rtDeclareVariable(float3,		shading_normal,		attribute shading_normal, );
rtDeclareVariable(float3,		texcoord,			attribute texcoord, );
rtDeclareVariable(optix::Ray,	ray,				rtCurrentRay, );
rtDeclareVariable(float,		t_hit,				rtIntersectionDistance, );

RT_PROGRAM void default_lit_closest_hit() //ray-type = 0(common_ray)
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;

	current_prd.seed += 197;
	float z1 = rnd(current_prd.seed);
	current_prd.seed += 197;
	float z2 = rnd(current_prd.seed);
	float3 baseColor;

	// initialize surface info
	SurfaceInfo IN;
	IN.baseColor = albedo;
	IN.transparent = transparent;
	IN.metallic = metallic;
	IN.smoothness = smoothness;
	IN.normal = ffnormal;

	current_prd.radiance = make_float3(0);

	{
		int in_to_out = dot(ray.direction, world_geometric_normal) > 0;

		float3 p;
		PerRayData_pathtrace prd;
		Ray next_ray;
		prd.depth = current_prd.depth + 1;
		prd.seed = current_prd.seed;
		prd.radiance = make_float3(0);

		float3 a;
		float b;
		baseColor = DiffuseAndSpecularFromMetallic(IN.baseColor, IN.metallic, a, b);

		if (current_prd.depth < 6) 
		{
			{
				float max_diffuse = max(max(baseColor.x, baseColor.y), baseColor.z);
				if (z1 < max_diffuse * transparent / 4) //͸�䲿��
				{
					if (refract(p, ray.direction, ffnormal, in_to_out ? 1.0f / refraction_index : refraction_index)) {
						prd.countEmitted = false;

						next_ray = make_Ray(hitpoint, p, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);

						rtTrace(top_object, next_ray, prd);

						current_prd.radiance += prd.radiance * baseColor / max_diffuse * 4;
					}
				}
				if (!in_to_out) {
					if (z2 < max_diffuse * (1 - transparent)) { //���䲿��
						cosine_sample_hemisphere(z1, z2, p);
						optix::Onb onb(ffnormal);
						onb.inverse_transform(p);

						prd.countEmitted = false;

						next_ray = make_Ray(hitpoint, p, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);

						rtTrace(top_object, next_ray, prd);
						current_prd.radiance += prd.radiance * baseColor / max_diffuse * M_1_PI;
					}
				}
			}
			if (z1 < 1.f / (current_prd.depth+3))
			{// ���䲿��
				float pd = M_PI;
				float3 n = make_float3(0,0,1);
				//uniform_sample_hemisphere(z1, z2, n);
				sample_GGX(make_float2(z1, z2), IN.smoothness, n, pd);
				if (pd != 0) {
					optix::Onb onb(ffnormal);
					onb.inverse_transform(n);
					p = reflect(ray.direction, n);

					if (dot(p, ffnormal) > 0) {
						prd.countEmitted = false;

						next_ray = make_Ray(hitpoint, p, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);

						rtTrace(top_object, next_ray, prd);

						current_prd.radiance += PBS(IN, p, prd.radiance, -ray.direction) * (current_prd.depth+3) / pd;
					}
				}
			}
		}
	}
	
	if (z1 > 1.f / (current_prd.depth + 1)) return;

	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = -dot(light.normal, L);

		// cast shadow ray
		if (nDl > 0.0f && LnDl > 0.0f)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = 1;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if (shadow_prd.inShadow != 0)
			{
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = LnDl * A / (M_PIf * Ldist * Ldist);
				float3 light_satu = light.emission * weight * shadow_prd.inShadow;
				current_prd.radiance += (PBS(IN, L, light_satu, -ray.direction) + nDl * LnDl * light_satu * baseColor) * (current_prd.depth + 1);
			}
		}
	}
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void default_lit_any_hit() //ray-type = 1(shadow_ray)
{
	if (transparent == 0) {
		current_prd_shadow.inShadow = 0;
		rtTerminateRay();
	}
	else {
		current_prd_shadow.inShadow *= transparent * 0.8;
		rtIgnoreIntersection();
	}
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
    current_prd.radiance = bg_color;
}


