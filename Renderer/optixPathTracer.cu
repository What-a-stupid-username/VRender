#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 radiance;
    unsigned int seed;
    int depth;
    int countEmitted;
};

struct PerRayData_pathtrace_shadow
{
    float inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  rnd_seed, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  common_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );

rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;


RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x + rnd_seed, frame_number);
    do 
    {
        //
        // Sample pixel using jittering
        //
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
		
		float3 ray_origin = eye/* + eye_jit*/;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.countEmitted = true;
        prd.seed = seed;
		prd.depth = 0;
		prd.radiance = make_float3(0);

        Ray ray = make_Ray(ray_origin, ray_direction, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_object, ray, prd);

        result += prd.radiance;

    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

	//output_buffer[launch_index] = make_float4(0,0,0,1);
    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,	  diffuse_color, , );
rtDeclareVariable(float,	  trans, , ) = 0;
rtDeclareVariable(float,	  spec, , ) = 0;
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );


RT_PROGRAM void diffuse()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    float z1=rnd(current_prd.seed);
	current_prd.seed += 197;
    float z2=rnd(current_prd.seed);

	{
		float3 p;
		PerRayData_pathtrace prd;
		Ray next_ray;
		prd.depth = current_prd.depth + 1;
		prd.seed = current_prd.seed;
		prd.radiance = make_float3(0);
		if (current_prd.depth < 3) {
			float max_diffuse = max(max(diffuse_color.x, diffuse_color.y), diffuse_color.z);
			if (z1 < max_diffuse * (1 - trans))
			{
				cosine_sample_hemisphere(z1, z2, p);
				optix::Onb onb(ffnormal);
				onb.inverse_transform(p);

				prd.countEmitted = false;

				next_ray = make_Ray(hitpoint, p, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);

				rtTrace(top_object, next_ray, prd);
				current_prd.radiance += prd.radiance * diffuse_color / max_diffuse * (1 - spec);
			}
			if (z1 < max(spec, 0.04)) {
				p = reflect(ray.direction, ffnormal);

				prd.countEmitted = true;

				next_ray = make_Ray(hitpoint, p, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);

				rtTrace(top_object, next_ray, prd);
				current_prd.radiance += prd.radiance;
			}
		}
		if (z1 < trans) {
			if (current_prd.depth < 9) {
				refract(p, ray.direction, ffnormal, 1.5);

				prd.countEmitted = true;

				next_ray = make_Ray(hitpoint, p, common_ray_type, scene_epsilon, RT_DEFAULT_MAX);

				rtTrace(top_object, next_ray, prd);
				current_prd.radiance += prd.radiance * 1;
			}
		}
	}
	


	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = -dot(light.normal, L);

		// cast shadow ray
		if (nDl > 0.0f && LnDl > 0.0f)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = 1;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if (shadow_prd.inShadow != 0)
			{
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				result += light.emission * weight * shadow_prd.inShadow;
			}
		}
	}
	current_prd.radiance += result * diffuse_color;
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
	if (trans == 0) {
		current_prd_shadow.inShadow = 0;
		rtTerminateRay();
	}
	else {
		current_prd_shadow.inShadow *= trans * 0.8;
		rtIgnoreIntersection();
	}
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
    current_prd.radiance = bg_color;
}


