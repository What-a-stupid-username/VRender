#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, plane, , );
rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, anchor, , );
rtDeclareVariable(int, lgt_instance, , ) = {0};

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx)
{
  float3 n = make_float3( plane );
  float dt = dot(ray.direction, n );
  float t = (plane.w - dot(n, ray.origin))/dt;
  if( t > ray.tmin && t < ray.tmax ) {
    float3 p = ray.origin + ray.direction * t;
    float3 vi = p - anchor;
    float a1 = dot(v1, vi);
    if(a1 >= 0 && a1 <= 1){
      float a2 = dot(v2, vi);
      if(a2 >= 0 && a2 <= 1){
        if( rtPotentialIntersection( t ) ) {
          shading_normal = geometric_normal = n;
          texcoord = make_float3(a1,a2,0);
          lgt_idx = lgt_instance;
          rtReportIntersection( 0 );
        }
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  // v1 and v2 are scaled by 1./length^2.  Rescale back to normal for the bounds computation.
  const float3 tv1  = v1 / dot( v1, v1 );
  const float3 tv2  = v2 / dot( v2, v2 );
  const float3 p00  = anchor;
  const float3 p01  = anchor + tv1;
  const float3 p10  = anchor + tv2;
  const float3 p11  = anchor + tv1 + tv2;
  const float  area = length(cross(tv1, tv2));
  
  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( p00, p01 ), fminf( p10, p11 ) );
    aabb->m_max = fmaxf( fmaxf( p00, p01 ), fmaxf( p10, p11 ) );
  } else {
    aabb->invalidate();
  }
}

